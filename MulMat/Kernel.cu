#include "hip/hip_runtime.h" 

#include <stdio.h>

typedef unsigned char uchar;

//	If you want to have the .cu intellisense:
//	Adding '.cu' under c++ extension in visual studio settings would enable syntax highlighting for c++ 
//	keywords only. EDIT: It in Tools -> Options -> Text Editor -> File Extension type in cu and select 
//	Microsoft Visual C++ as the editor and click add

/************************************************************************
// KERNEL qui permet de faire une multiplication scalaire d'une matrice
// d'entier. Chaque thread s'occupe d'un résultat
/***********************************************************************/
__global__
static void Kernel_ScalaireMulMat_Int(uchar *MatI, int K, uchar *MatO)
{
	int ImgNumColonne = blockIdx.x  * blockDim.x + threadIdx.x;
	int ImgNumLigne = blockIdx.y  * blockDim.y + threadIdx.y;
	int ImageWidth = blockDim.x * gridDim.x;
	int Index = ImgNumLigne * ImageWidth + ImgNumColonne;

	MatO[Index] = MatI[Index] + 50;
}

__global__
static void Kernel_Grad(uchar *MatI, int *gX, int *gY, int *MatO)
{

}

/************************************************************************
// Fonction de lancement du kernel qui permet de faire une multiplication
// scalaire d'une matrice d'entier.
/***********************************************************************/
extern "C" hipError_t Launcher_ScalaireMulMat_Int(uchar *pMatI, int K, uchar *pMatO, dim3 DimMat)
{
	//	1. Initialize data
	//	Choose which GPU to run on, change this on a multi-GPU system.    
	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return cudaStatus;
	}

	int BLOCK_SIZE = 16;
	uchar *gMatI, *gMatO;

	int gX[3][3] = { { -1, 0, 1 },
					{ -2, 0, 2 },
					{ -1, 0, 1 } };

	int gY[3][3] = { { -1, -2, -1 },
					{ 0, 0, 0 },
					{ 1, 2, 1 } };

	int size = (DimMat.y - 2) * (DimMat.x - 2);
	int *gGrad = new int[size];
	//	Grid of BLOCK_SIZE * BLOCK_SIZE blocks
	dim3 dimGrid(BLOCK_SIZE, BLOCK_SIZE);
	//	Block of BLOCK_SIZE * BLOCK_SIZE threads
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

	size_t memSize = DimMat.x * DimMat.y * sizeof(uchar);
	size_t memSizeInt = (DimMat.x - 2) * (DimMat.y - 2) * sizeof(int);
	size_t memSizeConv = 3 * 3 * sizeof(int);
	//	2. Allocate memory for the data on the GPU
	cudaStatus = hipMalloc(&gMatI, memSize);
	cudaStatus = hipMalloc(&gMatO, memSize);
	cudaStatus = hipMalloc(&gGrad, memSizeInt);

	//cudaStatus = cudaMalloc(&gX, memSizeConv);
	//cudaStatus = cudaMalloc(&gY, memSizeConv);
	//	3. Copy the data on the GPU
	cudaStatus = hipMemcpy(gMatI, pMatI, memSize, hipMemcpyHostToDevice);

	//	4. Launch kernel
	//Kernel_ScalaireMulMat_Int <<<dimGrid, dimBlock >>>(gMatI, K, gMatO);
//	Kernel_Grad << <dimGrid, dimBlock >> >(gMatI, gX, gY, gGrad);
	
	cudaStatus = hipDeviceSynchronize();	
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Kernel failed!");
		return cudaStatus;
	}

	//	5. Copy the data back on the CPU
	hipMemcpy(pMatO, gMatO, memSize, hipMemcpyDeviceToHost);

	//	6. Free the memory of the GPU
	hipFree(gMatI);
	hipFree(gMatO);

	return cudaStatus;
}
